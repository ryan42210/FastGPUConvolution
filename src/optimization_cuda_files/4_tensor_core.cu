#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <hip/hip_fp16.h>
#include <mma.h>

#include "gpu-new-forward.h"
using namespace nvcuda;

constexpr int TILE_HEIGHT = 8;
constexpr int TILE_WIDTH = 16;
constexpr int WARP_SIZE = 32;
constexpr int WMMA_M = 16;
constexpr int WMMA_N = 16;
constexpr int WMMA_K = 16;

__global__ void tensor_mul_kernel(float * __restrict__ output, const float * __restrict__ input, const float * __restrict__ mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S, const int H_out, const int W_out, const int unrolled_width, const int vector_len) {
    __shared__ __half tiled_unrolled[WMMA_K * WMMA_N];
    __shared__ __half tiled_mask[WMMA_M * WMMA_K];

    const int b = blockIdx.z;

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int w = blockDim.x * blockIdx.x + tx;
    const int h = TILE_WIDTH * blockIdx.y + ty;
    const int out_w = w % W_out;
    const int out_h = w / W_out;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major> mask_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major> unrolled_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, __half> acc_frag;

    wmma::fill_fragment(acc_frag, static_cast<__half>(0));

    for (int stage = 0; stage < ceil((float)vector_len / WMMA_K); stage++) {
        // load mask matrix
        if (stage * TILE_WIDTH + tx < vector_len) {
            if (h < M) {
                tiled_mask[ty * blockDim.x + tx] = __float2half(mask[h * vector_len + stage * WMMA_K + tx]);
            } else {
                tiled_mask[ty * blockDim.x + tx] = static_cast<__half>(0);
            }
            if (h + 8 < M) {
                tiled_mask[(ty + 8) * blockDim.x + tx] = __float2half(mask[(h + 8) * vector_len + stage * WMMA_K + tx]);
            } else {
                tiled_mask[(ty + 8) * blockDim.x + tx] = static_cast<__half>(0);
            }
        }

        if (w < unrolled_width) {
            int unrolled_h = (stage * TILE_WIDTH + ty);
            if (unrolled_h < vector_len) {
                int in_c = unrolled_h % (K * K);
                int in_w = (in_c % K) + out_w * S;
                int in_h = (in_c / K) + out_h * S;
                tiled_unrolled[ty * TILE_WIDTH + tx] = __float2half(input[b * (C * H * W) + (unrolled_h / (K * K)) * (H * W) + in_h * (W) + in_w]);
            } else {
                tiled_unrolled[ty * TILE_WIDTH + tx] = static_cast<__half>(0);
            }
            if (unrolled_h + 8 < vector_len) {
                int in_c = (unrolled_h + 8) % (K * K);
                int in_w = (in_c % K) + out_w * S;
                int in_h = (in_c / K) + out_h * S;
                tiled_unrolled[(ty + 8) * TILE_WIDTH + tx] = __float2half(input[b * (C * H * W) + ((unrolled_h + 8)/ (K * K)) * (H * W) + in_h * (W) + in_w]);
            } else {
                tiled_unrolled[(ty + 8) * TILE_WIDTH + tx] = static_cast<__half>(0);
            }
        }
        __syncthreads();

        if (tx + ty * TILE_WIDTH < WARP_SIZE) {
            // Load the inputs
            wmma::load_matrix_sync(mask_frag, tiled_mask, WMMA_K);
            wmma::load_matrix_sync(unrolled_frag, tiled_unrolled, WMMA_N);
            wmma::mma_sync(acc_frag, mask_frag, unrolled_frag, acc_frag);
        }
        __syncthreads();
    }

    if (tx + ty * blockDim.x < WARP_SIZE) {
        wmma::store_matrix_sync(tiled_unrolled, acc_frag, WMMA_N, wmma::mem_row_major);
    }
    __syncthreads();
    if (h < M && w < unrolled_width) {
        output[b * (M * unrolled_width) + h * (unrolled_width) + w] = __half2float(tiled_unrolled[ty * blockDim.x + tx]);
    }
    if (h + 8 < M && w < unrolled_width) {
        output[b * (M * unrolled_width) + (h + 8) * (unrolled_width) + w] = __half2float(tiled_unrolled[(ty + 8) * blockDim.x + tx]);
    }
}
	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    hipMalloc((void **)device_input_ptr, B * C * H * W * sizeof(float));
    hipMalloc((void **)device_output_ptr, B * M * H_out * W_out * sizeof(float));
    hipMalloc((void **)device_mask_ptr, M * C * K * K * sizeof(float));
    
    hipMemcpy((void *)(*device_mask_ptr), (const void *)host_mask, M * C * K * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void *)(*device_input_ptr), (const void *)host_input, B * C * H * W * sizeof(float), hipMemcpyHostToDevice);

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    const int out_w = H_out * W_out;

    dim3 mat_grid(ceil((float)out_w / TILE_WIDTH), ceil((float)M / TILE_WIDTH), B);
    dim3 mat_block(TILE_WIDTH, TILE_HEIGHT, 1);
    tensor_mul_kernel<<<mat_grid, mat_block>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S, H_out, W_out, out_w, C*K*K);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    // Copy the output back to host
    hipMemcpy(host_output, device_output, B * M * H_out * W_out * sizeof(float), hipMemcpyDeviceToHost);
   
    // Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}